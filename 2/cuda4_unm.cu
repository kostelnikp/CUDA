#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava, 2020/11
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage with unified memory.
//
// Image transformation from RGB to BW schema.
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include "cuda_img.h"

// Demo kernel to transform RGB color schema to BW schema
__global__ void kernel_grayscale(CudaImg t_color_cuda_img, CudaImg t_bw_cuda_img)
{
    // X,Y coordinates and check image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if (l_y >= t_color_cuda_img.m_size.y)
        return;
    if (l_x >= t_color_cuda_img.m_size.x)
        return;

    // Get point from color picture
    uchar3 l_bgr = t_color_cuda_img.m_p_uchar3[l_y * t_color_cuda_img.m_size.x + l_x];

    // Store BW point to new image
    t_bw_cuda_img.m_p_uchar1[l_y * t_bw_cuda_img.m_size.x + l_x].x = l_bgr.x * 0.11 + l_bgr.y * 0.59 + l_bgr.z * 0.30;
}

void cu_run_grayscale(CudaImg t_color_cuda_img, CudaImg t_bw_cuda_img)
{
    hipError_t l_cerr;

    // Grid creation, size of grid must be equal or greater than images
    int l_block_size = 16;
    dim3 l_blocks((t_color_cuda_img.m_size.x + l_block_size - 1) / l_block_size, (t_color_cuda_img.m_size.y + l_block_size - 1) / l_block_size);
    dim3 l_threads(l_block_size, l_block_size);
    kernel_grayscale<<<l_blocks, l_threads>>>(t_color_cuda_img, t_bw_cuda_img);

    if ((l_cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    hipDeviceSynchronize();
}

__global__ void kernel_maska(CudaImg t_color_cuda_img, CudaImg t_maska_cuda_img, uchar3 maska)
{
    int index_x = threadIdx.x + blockIdx.x * blockDim.x;
    int index_y = threadIdx.y + blockIdx.y * blockDim.y;

    if (index_x >= t_color_cuda_img.m_size.x)
        return;
    if (index_y >= t_color_cuda_img.m_size.y)
        return;

    *(t_maska_cuda_img.at3(index_y, index_x)) = *(t_color_cuda_img.at3(index_y, index_x));
    t_maska_cuda_img.at3(index_y, index_x)->x *= (float)maska.x / 255.0;
    t_maska_cuda_img.at3(index_y, index_x)->y *= (float)maska.y / 255.0;
    t_maska_cuda_img.at3(index_y, index_x)->z *= (float)maska.z / 255.0;
}

void cu_run_maska(CudaImg t_color_cuda_img, CudaImg t_maska_cuda_img, uchar3 maska)
{
    hipError_t l_cerr;

    int l_block_size = 32;
    dim3 l_blocks((t_color_cuda_img.m_size.x + l_block_size - 1) / l_block_size, (t_color_cuda_img.m_size.y + l_block_size - 1) / l_block_size);
    dim3 l_threads(l_block_size, l_block_size);

    kernel_maska<<<l_blocks, l_threads>>>(t_color_cuda_img, t_maska_cuda_img, maska);

    if ((l_cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    hipDeviceSynchronize();
}

__global__ void kernel_split(CudaImg t_color_cuda_img, CudaImg t_img1_cuda, CudaImg t_img2_cuda, bool flipX)
{
    int l_y = threadIdx.y + blockIdx.y * blockDim.y;
    int l_x = threadIdx.x + blockIdx.x * blockDim.x;
    if (l_x >= t_color_cuda_img.m_size.x)
        return;
    if (l_y >= t_color_cuda_img.m_size.y)
        return;

    int half_point = t_color_cuda_img.m_size.x / 2.0;

    if (flipX)
    {
        if (l_x >= half_point)
        {
            *(t_img2_cuda.at3((t_img2_cuda.m_size.x - 1) - (l_x - half_point), l_y)) = *(t_color_cuda_img.at3(l_x, l_y));
        }
        else
        {
            *(t_img1_cuda.at3((t_img1_cuda.m_size.x - 1) - l_x, l_y)) = *(t_color_cuda_img.at3(l_x, l_y));
        }
    }
    else
    {
        if (l_x >= half_point)
        {
            *(t_img2_cuda.at3((l_x - half_point), (t_img2_cuda.m_size.y - 1) - l_y)) = *(t_color_cuda_img.at3(l_x, l_y));
        }
        else
        {
            *(t_img1_cuda.at3(l_x, (t_img1_cuda.m_size.y - 1) - l_y)) = *(t_color_cuda_img.at3(l_x, l_y));
        }
    }
}

void cu_run_split(CudaImg t_color_cuda_img, CudaImg t_img1_cuda, CudaImg t_img2_cuda, bool flipX)
{
    hipError_t l_cerr;

    int l_block_size = 32;
    dim3 l_blocks((t_color_cuda_img.m_size.x + l_block_size - 1) / l_block_size, (t_color_cuda_img.m_size.y + l_block_size - 1) / l_block_size);
    dim3 l_threads(l_block_size, l_block_size);

    kernel_split<<<l_blocks, l_threads>>>(t_color_cuda_img, t_img1_cuda, t_img2_cuda, flipX);

    if ((l_cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    hipDeviceSynchronize();
}